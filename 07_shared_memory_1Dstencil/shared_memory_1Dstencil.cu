#include "hip/hip_runtime.h"
#include <stdio.h>
#define RADIUS 3;
#define BLOCK_SIZE 12;
#define ARRAY_SIZE 78;

int *array_of_ones(int size);
void print_array(int *array, int size) ;

__global__ void stencil_1d(int *in, int *out) {
  __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;

  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {
    temp[lindex - RADIUS] = in[gindex - RADIUS];
    temp[lindex - BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
  }

  __syncthreads();

  int result = 0;
  for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
    result += temp[lindex + offset];

  out[gindex] = result;
}

int main(void) {
  int thread_size = RADIUS * 2 + 1;
  int *in = array_of_ones(ARRAY_SIZE);
  int *out;
  int size = ARRAY_SIZE * sizeof(int);

  hipMalloc((void **)&d_a, ARRAY_SIZE);
  stencil_1d<<<BLOCK_SIZE, thread_size>>>(in, out);

  print_array(in, ARRAY_SIZE);
  print_array(out, ARRAY_SIZE);
  free(in); free(out);

  return 0;
}

void print_array(int *array, int size) {
  printf("\n");
  for (int i = 0; i < size; ++i) {
    printf("%d ", array[i]);
  }
  printf("\n");
}

int *array_of_ones(int size) {
  array = (int *)malloc(size);
  for (int i = 0; i < size; ++i) {
    array[i] = 1;
  }
  return array;
}
