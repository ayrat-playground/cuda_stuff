
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void array_of_ones(int *array, int size);
void print_array(int *array, int size);

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size); array_of_ones(a, N); print_array(a, N);
  b = (int *)malloc(size); array_of_ones(b, N); print_array(b, N);
  c = (int *)malloc(size);

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  add<<<1,N>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  print_array(c, N);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}

void array_of_ones(int *array, int size) {
  for (int i = 0; i < size; ++i) {
    array[i] = rand() % 10 + 1;
  }
}

void print_array(int *array, int size) {
  printf("\n");
  for (int i = 0; i < size; ++i) {
    printf("%d ", array[i]);
  }
  printf("\n");
}
