#include "hip/hip_runtime.h"
#include <stdio.h>
#define N (2048*2048)
#define THEADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

void array_of_ones(int *array, int size);
void print_array(int *array, int size);

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size); array_of_randoms(a, N); print_array(a, N);
  b = (int *)malloc(size); array_of_randoms(b, N); print_array(b, N);
  c = (int *)malloc(size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<(N + THEADS_PER_BLOCK - 1) / THEADS_PER_BLOCK,THEADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  print_array(c, N);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}

void array_of_randoms(int *array, int size) {
  for (int i = 0; i < size; ++i) {
    array[i] = rand() % 10 + 1;
  }
}

void print_array(int *array, int size) {
  printf("\n");
  for (int i = 0; i < size; ++i) {
    printf("%d ", array[i]);
  }
  printf("\n");
}
